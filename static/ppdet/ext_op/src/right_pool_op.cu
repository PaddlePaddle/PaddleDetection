#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/memory/memory.h"
#include <vector>
#include "util.cu.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaximumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaximumNumBlocks);
}

template <typename T>
class RightPoolOpCUDAKernel : public framework::OpKernel<T> {
public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");
    auto *x = ctx.Input<Tensor>("X");
    auto *max_map = ctx.Output<Tensor>("MaxMap");
    auto *output = ctx.Output<Tensor>("Output");
    auto *x_data = x->data<T>();
    auto x_dims = x->dims();
    int NC_num = x_dims[0] * x_dims[1];
    int height = x_dims[2];
    int width = x_dims[3];
    int num = x->numel();
    auto& dev_ctx = ctx.cuda_device_context();

    int *max_map_data = max_map->mutable_data<int>(x_dims, dev_ctx.GetPlace());
    T *output_data = output->mutable_data<T>(x_dims, dev_ctx.GetPlace());
    auto gpu_place = boost::get<platform::CUDAPlace>(dev_ctx.GetPlace());
    
    int threads = kNumCUDAThreads;
    int blocks = NumBlocks(num / width);
  
    auto max_val_ptr = memory::Alloc(gpu_place, num / width * sizeof(T));
    T* max_val_data = reinterpret_cast<T*>(max_val_ptr->ptr());
    auto max_ind_ptr = memory::Alloc(gpu_place, num / width * sizeof(int));
    int* max_ind_data = reinterpret_cast<int*>(max_ind_ptr->ptr());

    GetMaxInfo<T><<<blocks, threads, 0, dev_ctx.stream()>>>(x->data<T>(), NC_num, height, width, 3, false, max_val_data, max_ind_data, max_map_data);

    blocks = NumBlocks(num);
    ScatterAddFw<T><<<blocks, threads, 0, dev_ctx.stream()>>>(x->data<T>(), max_map_data, NC_num, height, width, 3, output_data);

  }
};

template <typename T>
class RightPoolGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* x = ctx.Input<Tensor>("X");
    auto* max_map = ctx.Input<Tensor>("MaxMap");
    auto* out_grad = ctx.Input<Tensor>(framework::GradVarName("Output"));
    auto* in_grad = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto x_dims = x->dims();
    
    auto& dev_ctx = ctx.cuda_device_context();
    T* in_grad_data = in_grad->mutable_data<T>(x_dims, dev_ctx.GetPlace());
    auto gpu_place = boost::get<platform::CUDAPlace>(dev_ctx.GetPlace());
    
    int threads = kNumCUDAThreads;
    int NC_num = x_dims[0] * x_dims[1];
    int height = x_dims[2];
    int width = x_dims[3];
    int grad_num = in_grad->numel();
    int blocks = NumBlocks(grad_num);
    FillConstant<T><<<blocks, threads, 0, dev_ctx.stream()>>>(in_grad_data, 0, grad_num);

    ScatterAddBw<T><<<blocks, threads, 0, dev_ctx.stream()>>>(out_grad->data<T>(), max_map->data<int>(), NC_num, height, width, 3, in_grad_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(right_pool,
                        ops::RightPoolOpCUDAKernel<float>,
                        ops::RightPoolOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(right_pool_grad,
                        ops::RightPoolGradOpCUDAKernel<float>,
                        ops::RightPoolGradOpCUDAKernel<double>);
