#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */


#include <cassert>
#include <cmath>

#ifdef __HIPCC__
// Designates functions callable from the host (CPU) and the device (GPU)
#define HOST_DEVICE __host__ __device__
#define HOST_DEVICE_INLINE HOST_DEVICE __forceinline__
#else
#include <algorithm>
#define HOST_DEVICE
#define HOST_DEVICE_INLINE HOST_DEVICE inline
#endif

#include "paddle/extension.h"

#include <vector>

namespace {

template <typename T>
struct RotatedBox {
  T x_ctr, y_ctr, w, h, a;
};

template <typename T>
struct Point {
  T x, y;
  HOST_DEVICE_INLINE Point(const T& px = 0, const T& py = 0) : x(px), y(py) {}
  HOST_DEVICE_INLINE Point operator+(const Point& p) const {
    return Point(x + p.x, y + p.y);
  }
  HOST_DEVICE_INLINE Point& operator+=(const Point& p) {
    x += p.x;
    y += p.y;
    return *this;
  }
  HOST_DEVICE_INLINE Point operator-(const Point& p) const {
    return Point(x - p.x, y - p.y);
  }
  HOST_DEVICE_INLINE Point operator*(const T coeff) const {
    return Point(x * coeff, y * coeff);
  }
};

template <typename T>
HOST_DEVICE_INLINE T dot_2d(const Point<T>& A, const Point<T>& B) {
  return A.x * B.x + A.y * B.y;
}

template <typename T>
HOST_DEVICE_INLINE T cross_2d(const Point<T>& A, const Point<T>& B) {
  return A.x * B.y - B.x * A.y;
}

template <typename T>
HOST_DEVICE_INLINE void get_rotated_vertices(
    const RotatedBox<T>& box,
    Point<T> (&pts)[4]) {
  // M_PI / 180. == 0.01745329251
  //double theta = box.a * 0.01745329251;
  //MODIFIED
  double theta = box.a;
  T cosTheta2 = (T)cos(theta) * 0.5f;
  T sinTheta2 = (T)sin(theta) * 0.5f;

  // y: top --> down; x: left --> right
  pts[0].x = box.x_ctr - sinTheta2 * box.h - cosTheta2 * box.w;
  pts[0].y = box.y_ctr + cosTheta2 * box.h - sinTheta2 * box.w;
  pts[1].x = box.x_ctr + sinTheta2 * box.h - cosTheta2 * box.w;
  pts[1].y = box.y_ctr - cosTheta2 * box.h - sinTheta2 * box.w;
  pts[2].x = 2 * box.x_ctr - pts[0].x;
  pts[2].y = 2 * box.y_ctr - pts[0].y;
  pts[3].x = 2 * box.x_ctr - pts[1].x;
  pts[3].y = 2 * box.y_ctr - pts[1].y;
}

template <typename T>
HOST_DEVICE_INLINE int get_intersection_points(
    const Point<T> (&pts1)[4],
    const Point<T> (&pts2)[4],
    Point<T> (&intersections)[24]) {
  // Line vector
  // A line from p1 to p2 is: p1 + (p2-p1)*t, t=[0,1]
  Point<T> vec1[4], vec2[4];
  for (int i = 0; i < 4; i++) {
    vec1[i] = pts1[(i + 1) % 4] - pts1[i];
    vec2[i] = pts2[(i + 1) % 4] - pts2[i];
  }

  // Line test - test all line combos for intersection
  int num = 0; // number of intersections
  for (int i = 0; i < 4; i++) {
    for (int j = 0; j < 4; j++) {
      // Solve for 2x2 Ax=b
      T det = cross_2d<T>(vec2[j], vec1[i]);

      // This takes care of parallel lines
      if (fabs(det) <= 1e-14) {
        continue;
      }

      auto vec12 = pts2[j] - pts1[i];

      T t1 = cross_2d<T>(vec2[j], vec12) / det;
      T t2 = cross_2d<T>(vec1[i], vec12) / det;

      if (t1 >= 0.0f && t1 <= 1.0f && t2 >= 0.0f && t2 <= 1.0f) {
        intersections[num++] = pts1[i] + vec1[i] * t1;
      }
    }
  }

  // Check for vertices of rect1 inside rect2
  {
    const auto& AB = vec2[0];
    const auto& DA = vec2[3];
    auto ABdotAB = dot_2d<T>(AB, AB);
    auto ADdotAD = dot_2d<T>(DA, DA);
    for (int i = 0; i < 4; i++) {
      // assume ABCD is the rectangle, and P is the point to be judged
      // P is inside ABCD iff. P's projection on AB lies within AB
      // and P's projection on AD lies within AD

      auto AP = pts1[i] - pts2[0];

      auto APdotAB = dot_2d<T>(AP, AB);
      auto APdotAD = -dot_2d<T>(AP, DA);

      if ((APdotAB >= 0) && (APdotAD >= 0) && (APdotAB <= ABdotAB) &&
          (APdotAD <= ADdotAD)) {
        intersections[num++] = pts1[i];
      }
    }
  }

  // Reverse the check - check for vertices of rect2 inside rect1
  {
    const auto& AB = vec1[0];
    const auto& DA = vec1[3];
    auto ABdotAB = dot_2d<T>(AB, AB);
    auto ADdotAD = dot_2d<T>(DA, DA);
    for (int i = 0; i < 4; i++) {
      auto AP = pts2[i] - pts1[0];

      auto APdotAB = dot_2d<T>(AP, AB);
      auto APdotAD = -dot_2d<T>(AP, DA);

      if ((APdotAB >= 0) && (APdotAD >= 0) && (APdotAB <= ABdotAB) &&
          (APdotAD <= ADdotAD)) {
        intersections[num++] = pts2[i];
      }
    }
  }

  return num;
}

template <typename T>
HOST_DEVICE_INLINE int convex_hull_graham(
    const Point<T> (&p)[24],
    const int& num_in,
    Point<T> (&q)[24],
    bool shift_to_zero = false) {
  assert(num_in >= 2);

  // Step 1:
  // Find point with minimum y
  // if more than 1 points have the same minimum y,
  // pick the one with the minimum x.
  int t = 0;
  for (int i = 1; i < num_in; i++) {
    if (p[i].y < p[t].y || (p[i].y == p[t].y && p[i].x < p[t].x)) {
      t = i;
    }
  }
  auto& start = p[t]; // starting point

  // Step 2:
  // Subtract starting point from every points (for sorting in the next step)
  for (int i = 0; i < num_in; i++) {
    q[i] = p[i] - start;
  }

  // Swap the starting point to position 0
  auto tmp = q[0];
  q[0] = q[t];
  q[t] = tmp;

  // Step 3:
  // Sort point 1 ~ num_in according to their relative cross-product values
  // (essentially sorting according to angles)
  // If the angles are the same, sort according to their distance to origin
  T dist[24];
  for (int i = 0; i < num_in; i++) {
    dist[i] = dot_2d<T>(q[i], q[i]);
  }

#ifdef __HIPCC__
  // CUDA version
  // In the future, we can potentially use thrust
  // for sorting here to improve speed (though not guaranteed)
  for (int i = 1; i < num_in - 1; i++) {
    for (int j = i + 1; j < num_in; j++) {
      T crossProduct = cross_2d<T>(q[i], q[j]);
      if ((crossProduct < -1e-6) ||
          (fabs(crossProduct) < 1e-6 && dist[i] > dist[j])) {
        auto q_tmp = q[i];
        q[i] = q[j];
        q[j] = q_tmp;
        auto dist_tmp = dist[i];
        dist[i] = dist[j];
        dist[j] = dist_tmp;
      }
    }
  }
#else
  // CPU version
  std::sort(
      q + 1, q + num_in, [](const Point<T>& A, const Point<T>& B) -> bool {
        T temp = cross_2d<T>(A, B);
        if (fabs(temp) < 1e-6) {
          return dot_2d<T>(A, A) < dot_2d<T>(B, B);
        } else {
          return temp > 0;
        }
      });
#endif

  // Step 4:
  // Make sure there are at least 2 points (that don't overlap with each other)
  // in the stack
  int k; // index of the non-overlapped second point
  for (k = 1; k < num_in; k++) {
    if (dist[k] > 1e-8) {
      break;
    }
  }
  if (k == num_in) {
    // We reach the end, which means the convex hull is just one point
    q[0] = p[t];
    return 1;
  }
  q[1] = q[k];
  int m = 2; // 2 points in the stack
  // Step 5:
  // Finally we can start the scanning process.
  // When a non-convex relationship between the 3 points is found
  // (either concave shape or duplicated points),
  // we pop the previous point from the stack
  // until the 3-point relationship is convex again, or
  // until the stack only contains two points
  for (int i = k + 1; i < num_in; i++) {
    while (m > 1 && cross_2d<T>(q[i] - q[m - 2], q[m - 1] - q[m - 2]) >= 0) {
      m--;
    }
    q[m++] = q[i];
  }

  // Step 6 (Optional):
  // In general sense we need the original coordinates, so we
  // need to shift the points back (reverting Step 2)
  // But if we're only interested in getting the area/perimeter of the shape
  // We can simply return.
  if (!shift_to_zero) {
    for (int i = 0; i < m; i++) {
      q[i] += start;
    }
  }

  return m;
}

template <typename T>
HOST_DEVICE_INLINE T polygon_area(const Point<T> (&q)[24], const int& m) {
  if (m <= 2) {
    return 0;
  }

  T area = 0;
  for (int i = 1; i < m - 1; i++) {
    area += fabs(cross_2d<T>(q[i] - q[0], q[i + 1] - q[0]));
  }

  return area / 2.0;
}

template <typename T>
HOST_DEVICE_INLINE T rboxes_intersection(
    const RotatedBox<T>& box1,
    const RotatedBox<T>& box2) {
  // There are up to 4 x 4 + 4 + 4 = 24 intersections (including dups) returned
  // from rotated_rect_intersection_pts
  Point<T> intersectPts[24], orderedPts[24];

  Point<T> pts1[4];
  Point<T> pts2[4];
  get_rotated_vertices<T>(box1, pts1);
  get_rotated_vertices<T>(box2, pts2);

  int num = get_intersection_points<T>(pts1, pts2, intersectPts);

  if (num <= 2) {
    return 0.0;
  }

  // Convex Hull to order the intersection points in clockwise order and find
  // the contour area.
  int num_convex = convex_hull_graham<T>(intersectPts, num, orderedPts, true);
  return polygon_area<T>(orderedPts, num_convex);
}

} // namespace

template <typename T>
HOST_DEVICE_INLINE T
rbox_iou_single(T const* const box1_raw, T const* const box2_raw) {
  // shift center to the middle point to achieve higher precision in result
  RotatedBox<T> box1, box2;
  auto center_shift_x = (box1_raw[0] + box2_raw[0]) / 2.0;
  auto center_shift_y = (box1_raw[1] + box2_raw[1]) / 2.0;
  box1.x_ctr = box1_raw[0] - center_shift_x;
  box1.y_ctr = box1_raw[1] - center_shift_y;
  box1.w = box1_raw[2];
  box1.h = box1_raw[3];
  box1.a = box1_raw[4];
  box2.x_ctr = box2_raw[0] - center_shift_x;
  box2.y_ctr = box2_raw[1] - center_shift_y;
  box2.w = box2_raw[2];
  box2.h = box2_raw[3];
  box2.a = box2_raw[4];

  const T area1 = box1.w * box1.h;
  const T area2 = box2.w * box2.h;
  if (area1 < 1e-14 || area2 < 1e-14) {
    return 0.f;
  }

  const T intersection = rboxes_intersection<T>(box1, box2);
  const T iou = intersection / (area1 + area2 - intersection);
  return iou;
}


// 2D block with 32 * 16 = 512 threads per block
const int BLOCK_DIM_X = 32;
const int BLOCK_DIM_Y = 16;

/**
   Computes ceil(a / b)
*/
template <typename T>
__host__ __device__ __forceinline__ T CeilDiv0(T a, T b) {
  return (a + b - 1) / b;
}

static inline int CeilDiv(const int a, const int b) {
  return (a + b -1)  / b;
}

template <typename T>
__global__ void rbox_iou_cuda_kernel(
    const int rbox1_num,
    const int rbox2_num,
    const T* rbox1_data_ptr,
    const T* rbox2_data_ptr,
    T* output_data_ptr) {

  // get row_start and col_start
  const int rbox1_block_idx = blockIdx.x * blockDim.x;
  const int rbox2_block_idx = blockIdx.y * blockDim.y;

  const int rbox1_thread_num = min(rbox1_num - rbox1_block_idx, blockDim.x);
  const int rbox2_thread_num = min(rbox2_num - rbox2_block_idx, blockDim.y);

  __shared__ T block_boxes1[BLOCK_DIM_X * 5];
  __shared__ T block_boxes2[BLOCK_DIM_Y * 5];


  // It's safe to copy using threadIdx.x since BLOCK_DIM_X >= BLOCK_DIM_Y
  if (threadIdx.x < rbox1_thread_num && threadIdx.y == 0) {
    block_boxes1[threadIdx.x * 5 + 0] =
        rbox1_data_ptr[(rbox1_block_idx + threadIdx.x) * 5 + 0];
    block_boxes1[threadIdx.x * 5 + 1] =
        rbox1_data_ptr[(rbox1_block_idx + threadIdx.x) * 5 + 1];
    block_boxes1[threadIdx.x * 5 + 2] =
        rbox1_data_ptr[(rbox1_block_idx + threadIdx.x) * 5 + 2];
    block_boxes1[threadIdx.x * 5 + 3] =
        rbox1_data_ptr[(rbox1_block_idx + threadIdx.x) * 5 + 3];
    block_boxes1[threadIdx.x * 5 + 4] =
        rbox1_data_ptr[(rbox1_block_idx + threadIdx.x) * 5 + 4];
  }

  // threadIdx.x < BLOCK_DIM_Y=rbox2_thread_num, just use same condition as above: threadIdx.y == 0
  if (threadIdx.x < rbox2_thread_num && threadIdx.y == 0) {
    block_boxes2[threadIdx.x * 5 + 0] =
        rbox2_data_ptr[(rbox2_block_idx + threadIdx.x) * 5 + 0];
    block_boxes2[threadIdx.x * 5 + 1] =
        rbox2_data_ptr[(rbox2_block_idx + threadIdx.x) * 5 + 1];
    block_boxes2[threadIdx.x * 5 + 2] =
        rbox2_data_ptr[(rbox2_block_idx + threadIdx.x) * 5 + 2];
    block_boxes2[threadIdx.x * 5 + 3] =
        rbox2_data_ptr[(rbox2_block_idx + threadIdx.x) * 5 + 3];
    block_boxes2[threadIdx.x * 5 + 4] =
        rbox2_data_ptr[(rbox2_block_idx + threadIdx.x) * 5 + 4];
  }

  // sync
  __syncthreads();

  if (threadIdx.x < rbox1_thread_num && threadIdx.y < rbox2_thread_num) {
    int offset = (rbox1_block_idx + threadIdx.x) * rbox2_num + rbox2_block_idx + threadIdx.y;
    output_data_ptr[offset] = rbox_iou_single<T>(block_boxes1 + threadIdx.x * 5, block_boxes2 + threadIdx.y * 5);
  }
}

#define CHECK_INPUT_GPU(x) PD_CHECK(x.place() == paddle::PlaceType::kGPU, #x " must be a GPU Tensor.")

std::vector<paddle::Tensor> RboxIouCUDAForward(const paddle::Tensor& rbox1, const paddle::Tensor& rbox2) {
    CHECK_INPUT_GPU(rbox1);
    CHECK_INPUT_GPU(rbox2);

    auto rbox1_num = rbox1.shape()[0];
    auto rbox2_num = rbox2.shape()[0];

    auto output = paddle::Tensor(paddle::PlaceType::kGPU);
    output.reshape({rbox1_num, rbox2_num});

    const int blocks_x = CeilDiv(rbox1_num, BLOCK_DIM_X);
    const int blocks_y = CeilDiv(rbox2_num, BLOCK_DIM_Y);

    dim3 blocks(blocks_x, blocks_y);
    dim3 threads(BLOCK_DIM_X, BLOCK_DIM_Y);

    PD_DISPATCH_FLOATING_TYPES(
        rbox1.type(),
        "rbox_iou_cuda_kernel",
        ([&] {
            rbox_iou_cuda_kernel<data_t><<<blocks, threads, 0, rbox1.stream()>>>(
                rbox1_num,
                rbox2_num,
                rbox1.data<data_t>(),
                rbox2.data<data_t>(),
                output.mutable_data<data_t>());
        }));

    return {output};
}


template <typename T>
void rbox_iou_cpu_kernel(
    const int rbox1_num,
    const int rbox2_num,
    const T* rbox1_data_ptr,
    const T* rbox2_data_ptr,
    T* output_data_ptr) {

    int i, j;
    for (i = 0; i < rbox1_num; i++) {
        for (j = 0; j < rbox2_num; j++) {
		int offset = i * rbox2_num + j;
		output_data_ptr[offset] = rbox_iou_single<T>(rbox1_data_ptr + i * 5, rbox2_data_ptr + j * 5);
        }
    }
}


#define CHECK_INPUT_CPU(x) PD_CHECK(x.place() == paddle::PlaceType::kCPU, #x " must be a CPU Tensor.")

std::vector<paddle::Tensor> RboxIouCPUForward(const paddle::Tensor& rbox1, const paddle::Tensor& rbox2) {
    CHECK_INPUT_CPU(rbox1);
    CHECK_INPUT_CPU(rbox2);

    auto rbox1_num = rbox1.shape()[0];
    auto rbox2_num = rbox2.shape()[0];

    auto output = paddle::Tensor(paddle::PlaceType::kCPU);
    output.reshape({rbox1_num, rbox2_num});

    PD_DISPATCH_FLOATING_TYPES(
        rbox1.type(),
        "rbox_iou_cpu_kernel",
        ([&] {
            rbox_iou_cpu_kernel<data_t>(
                rbox1_num,
                rbox2_num,
                rbox1.data<data_t>(),
                rbox2.data<data_t>(),
                output.mutable_data<data_t>());
        }));
    
    return {output};
}
